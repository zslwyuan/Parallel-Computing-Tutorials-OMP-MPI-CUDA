#include "hip/hip_runtime.h"
/*
 * This is a CUDA version of bellman_ford algorithm (Heuristic Variant)
 * Compile: nvcc -std=c++11 -arch=sm_52 -o cuda_bellman_ford cuda_bellman_ford.cu
 * Run: ./cuda_bellman_ford <input file> <number of blocks per grid> <number of threads per block>, you will find the output file 'output.txt'
 * */

#include <string>
#include <cassert>
#include <iostream>
#include <fstream>
#include <algorithm>
#include <iomanip>
#include <cstring>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <>

using std::string;
using std::cout;
using std::endl;

#define INF 1000000

/*
 * This is a CHECK function to check CUDA calls
 */
#define CHECK(call)                                                            \
		{                                                                              \
	const hipError_t error = call;                                            \
	if (error != hipSuccess)                                                  \
	{                                                                          \
		fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
		fprintf(stderr, "code: %d, reason: %s\n", error,                       \
				hipGetErrorString(error));                                    \
				exit(1);                                                               \
	}                                                                          \
		}


/**
 * utils is a namespace for utility functions
 * including I/O (read input file and print results) and matrix dimension convert(2D->1D) function
 */
namespace utils {
int N; //number of vertices
int *mat; // the adjacency matrix

void abort_with_error_message(string msg) {
	std::cerr << msg << endl;
	abort();
}

//translate 2-dimension coordinate to 1-dimension
int convert_dimension_2D_1D(int x, int y, int n) {
	return x * n + y;
}

int read_file(string filename) {
	std::ifstream inputf(filename, std::ifstream::in);
	if (!inputf.good()) {
		abort_with_error_message("ERROR OCCURRED WHILE READING INPUT FILE");
	}
	inputf >> N;
	//input matrix should be smaller than 20MB * 20MB (400MB, we don't have too much memory for multi-processors)
	assert(N < (1024 * 1024 * 20));
	mat = (int *) malloc(N * N * sizeof(int));
//        printf("%llu\n",(unsigned long long)mat);
	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++) {
			inputf >> mat[convert_dimension_2D_1D(i, j, N)];
		}
	return 0;
}

int print_result(bool has_negative_cycle, int *dist) {
	std::ofstream outputf("output.txt", std::ofstream::out);
	if (!has_negative_cycle) {
		for (int i = 0; i < N; i++) {
			if (dist[i] > INF)
				dist[i] = INF;
			outputf << dist[i] << '\n';
		}
		outputf.flush();
	} else {
		outputf << "FOUND NEGATIVE CYCLE!" << endl;
	}
	outputf.close();
	return 0;
}
}//namespace utils

// you may add some helper/kernel functions here.

//__constant__ int d_mat[10000*10000]:

__global__ void ClearChange(int N, int s, bool *d_has_change) 
{
	d_has_change[blockIdx.x]=0;
}
__global__ void UpdatePath(int N, int *mat, int *d_all_dist, bool *d_has_change) {

    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int elementSkip = blockDim.x * gridDim.x;
	int i,t;
    __shared__ int local_dist[10240];
    int local_mat;
	__shared__  bool local_has_change[1024];

	local_has_change[threadIdx.x] = 0;
	int local_dist_now;
    int weight;

	for (i = tid; i < N; i += elementSkip) 
	{
	    	local_dist[i]=d_all_dist[i];    
	    	__syncthreads();
	 }

	int now;
	for (now = 0; now < N; now++)
	{
		local_dist_now = d_all_dist[now];
		for (i = tid; i < N; i += elementSkip)
		{
			local_mat=mat[now*N+i];
			__syncthreads();
			weight = local_mat+local_dist_now;  
			if (weight<local_dist[i])
			{
				if (local_mat<INF&&local_dist_now<INF) local_dist[i] = weight;
				local_has_change[threadIdx.x] |= 1;
			}
		}
	}
	for (t = blockDim.x / 2; t > 0; t /= 2) 
	{
		if (threadIdx.x < t ) 
		{
		    local_has_change[threadIdx.x] |= local_has_change[threadIdx.x + t];
		}
		__syncthreads();
	}

	for (i = tid; i < N; i += elementSkip) 
	{
	    d_all_dist[i]=local_dist[i];    
	    __syncthreads();
	}
    if (threadIdx.x == 0) d_has_change[blockIdx.x]|= local_has_change[0];
}


/**
 * Bellman-Ford algorithm. Find the shortest path from vertex 0 to other vertices.
 * @param blockPerGrid number of blocks per grid
 * @param threadsPerBlock number of threads per block
 * @param n input size
 * @param *mat input adjacency matrix
 * @param *dist distance array
 * @param *has_negative_cycle a bool variable to recode if there are negative cycles
 */
void bellman_ford(int blocksPerGrid, int threadsPerBlock, int n, int *mat, int *dist, bool *has_negative_cycle) {


    int *all_dist = dist;

    dim3 blocks(blocksPerGrid);
    dim3 threads(threadsPerBlock);


    //allocate memory
    int *d_all_dist;
    bool  *d_has_change;
    bool has_change[64];
    int *d_mat;hipMalloc((void **)&d_mat, sizeof(int) * n * n);
    hipMalloc((void **)&d_has_change, sizeof(bool) * blocksPerGrid);
    hipMalloc((void **)&d_all_dist, sizeof(int) * n);
    //initialization and copy data from host to device
    for (int i = 0; i < n; i++) {
        all_dist[i] = INF;
    }
    for (int i = 0; i< blocksPerGrid; i++) has_change[i]=0;
    all_dist[0] = 0;

    hipMemcpy(d_mat, mat, sizeof(int)*n*n , hipMemcpyHostToDevice);

    hipMemcpy(d_all_dist, all_dist, sizeof(int) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_has_change, has_change, sizeof(bool) * blocksPerGrid, hipMemcpyHostToDevice);
    int iter = 0, s=0,c=0;


    bool global_change = 0;
    int dist0;int tj = 0;
    for (iter = 0; iter < n+100; iter++) {

    	global_change = 0;
        UpdatePath << < blocks, threads >> > (n,  d_mat, d_all_dist, d_has_change);
        hipMemcpy(has_change, d_has_change, sizeof(bool) * blocksPerGrid, hipMemcpyDeviceToHost);
        for (c = 0; c < blocksPerGrid; c+=1)global_change |= has_change[c];
       	ClearChange << < blocks, threads >> > (n, s, d_has_change);

       	if (iter>7)hipMemcpy(&dist0, d_all_dist, sizeof(int), hipMemcpyDeviceToHost);
        if (!global_change||dist0<0) {break;}
    }

    if (dist0<0||iter==n+100) *has_negative_cycle = 1;

    //copy results from device to host
    hipMemcpy(all_dist, d_all_dist, sizeof(int) * n, hipMemcpyDeviceToHost);

    //free memory

    hipFree(d_mat);
    hipFree(d_all_dist);
    hipFree(d_has_change);
}

int main(int argc, char **argv) {
	if (argc <= 1) {
		utils::abort_with_error_message("INPUT FILE WAS NOT FOUND!");
	}
	if (argc <= 3) {
		utils::abort_with_error_message("blocksPerGrid or threadsPerBlock WAS NOT FOUND!");
	}

	string filename = argv[1];
	int blockPerGrid = atoi(argv[2]);
	int threadsPerBlock = atoi(argv[3]);

	int *dist;
	bool has_negative_cycle = false;


	assert(utils::read_file(filename) == 0);
	dist = (int *) calloc(sizeof(int), utils::N);


	//time counter
	timeval start_wall_time_t, end_wall_time_t;
	float ms_wall;
	hipDeviceReset();
	//start timer
	gettimeofday(&start_wall_time_t, nullptr);
	//bellman-ford algorithm
	bellman_ford(blockPerGrid, threadsPerBlock, utils::N, utils::mat, dist, &has_negative_cycle);
	CHECK(hipDeviceSynchronize());
	//end timer
	gettimeofday(&end_wall_time_t, nullptr);
	ms_wall = ((end_wall_time_t.tv_sec - start_wall_time_t.tv_sec) * 1000 * 1000
			+ end_wall_time_t.tv_usec - start_wall_time_t.tv_usec) / 1000.0;

	std::cerr.setf(std::ios::fixed);
	std::cerr << std::setprecision(6) << "Time(s): " << (ms_wall/1000.0) << endl;
	utils::print_result(has_negative_cycle, dist);
	free(dist);
	free(utils::mat);

	return 0;
}
